#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <stdbool.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"


const int SCREEN_WIDTH = 300;
const int SCREEN_HEIGHT = 300;
const int BLOCKSIZE = 8;
const float stepSize = 1;

typedef struct Vec3f {
	float x;
	float y;
	float z;
} Vec3f_t;

typedef struct Ray {
	Vec3f_t pos;
	Vec3f_t dir;
	bool colored;
};

typedef struct dims {
	int width;
	int height;
} Dims;

typedef unsigned char Rgb[3];

SDL_Window* gWindow = NULL;
SDL_Renderer* gRenderer = NULL;

Vec3f_t cameraPos = { 0, 0, 0 };
Vec3f_t cameraDir = { 0, -1, 0 };
Vec3f_t cameraUp = { 0, 0, -1 };
Vec3f_t cameraRight = { -1, 0, 0 };

Vec3f_t blackHolePos = { 0, -20, 0 };

Ray rays[SCREEN_HEIGHT * SCREEN_WIDTH];
Rgb* frameBuffer = new Rgb[SCREEN_WIDTH * SCREEN_HEIGHT];
uint8_t* rgb_image;

float starFieldRadius = 80;
float fov = 100;

int width, height, bpp;


bool init();

void close();
void init_rays();
__global__ void propRays(Rgb* gpu_frameBuffer, uint8_t* gpu_rgb_image, Ray* gpu_rays, Dims* dims, Vec3f_t* gpu_blackHole);
void rodriguesFormula(Vec3f_t* rotVec, Vec3f_t v, Vec3f_t k, float theta);



int main(int argc, char** argv){

	Rgb* gpu_frameBuffer;
	uint8_t* gpu_rgb_image;
	Ray* gpu_rays;
	Dims* dims;
	Vec3f_t* gpu_blackHole;

	rgb_image = stbi_load("starmap_2020_4k_brighter.png", &width, &height, &bpp, 3);

	// Allocate space for the frameBuffer on the GPU
	if (hipMalloc(&gpu_frameBuffer, sizeof(Rgb)*SCREEN_HEIGHT*SCREEN_WIDTH) != hipSuccess) {
		fprintf(stderr, "Failed to allocate frameBuffer on GPU\n");
		exit(2);
	}

	// Allocate space for the rgb_image on the GPU
	if (hipMalloc(&gpu_rgb_image, sizeof(uint8_t) * width * height * 3) != hipSuccess) {
		fprintf(stderr, "Failed to allocate rgb_image on GPU\n");
		exit(2);
	}

	// Allocate space for the rays on the GPU
	if (hipMalloc(&gpu_rays, sizeof(Ray) * SCREEN_HEIGHT * SCREEN_WIDTH) != hipSuccess) {
		fprintf(stderr, "Failed to allocate rays on GPU\n");
		exit(2);
	}

	// Allocate space for the rays on the GPU
	if (hipMalloc(&dims, sizeof(Dims)) != hipSuccess) {
		fprintf(stderr, "Failed to allocate dims on GPU\n");
		exit(2);
	}

	// Allocate space for the frameBuffer on the GPU
	if (hipMalloc(&gpu_blackHole, sizeof(Vec3f_t)) != hipSuccess) {
		fprintf(stderr, "Failed to allocate blackHole on GPU\n");
		exit(2);
	}

	if (!rgb_image) {
		fprintf(stderr, "Cannot load file image %s\nSTB Reason: %s\n", "starmap_2020_4k_brighter.png", stbi_failure_reason());
		exit(0);
	}

	// Copy the cpu's rgb_image to the gpu with hipMemcpy
	if (hipMemcpy(gpu_rgb_image, rgb_image, sizeof(uint8_t) * width * height * 3, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy rgb_image to the GPU\n");
		exit(2);
	}

	// Copy the cpu's width to the gpu with hipMemcpy
	if (hipMemcpy(&dims->width, &width, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy width to the GPU\n");
		exit(2);
	}

	// Copy the cpu's height to the gpu with hipMemcpy
	if (hipMemcpy(&dims->height, &height, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy height to the GPU\n");
		exit(2);
	}

	// Copy the cpu's height to the gpu with hipMemcpy
	if (hipMemcpy(gpu_blackHole, &blackHolePos, sizeof(Vec3f_t), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy blackHole to the GPU\n");
		exit(2);
	}


	if (!init()) {
		printf("Failed to initialize!\n");
	}
	else {
		init_rays();

		// Copy the cpu's rays to the gpu with hipMemcpy
		if (hipMemcpy(gpu_rays, rays, sizeof(Ray) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyHostToDevice) != hipSuccess) {
			fprintf(stderr, "Failed to copy rays to the GPU\n");
		}

		bool quit = false;
		SDL_Event e;
		while (!quit) {
			while (SDL_PollEvent(&e) != 0) {
				if (e.type == SDL_QUIT) {
					quit = true;
				}
				else if (e.type == SDL_KEYDOWN)
				{
					//Select surfaces based on key press
					switch (e.key.keysym.sym)
					{
					case SDLK_UP:
						rodriguesFormula(&cameraDir, cameraDir, cameraRight, -0.1);
						rodriguesFormula(&cameraUp, cameraUp, cameraRight, -0.1);
						break;

					case SDLK_DOWN:
						rodriguesFormula(&cameraDir, cameraDir, cameraRight, 0.1);
						rodriguesFormula(&cameraUp, cameraUp, cameraRight, 0.1);
						break;

					case SDLK_LEFT:
						rodriguesFormula(&cameraDir, cameraDir, cameraUp, -0.1);
						rodriguesFormula(&cameraRight, cameraRight, cameraUp, -0.1);
						break;

					case SDLK_RIGHT:
						rodriguesFormula(&cameraDir, cameraDir, cameraUp, 0.1);
						rodriguesFormula(&cameraRight, cameraRight, cameraUp, 0.1);
						break;
					}
					init_rays();

					// Copy the cpu's rays to the gpu with hipMemcpy
					if (hipMemcpy(gpu_rays, rays, sizeof(Ray) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyHostToDevice) != hipSuccess) {
						fprintf(stderr, "Failed to copy rays to the GPU\n");
					}
				}
			}
			


			SDL_SetRenderDrawColor(gRenderer, 0x00, 0x00, 0x00, 0x00);
			SDL_RenderClear(gRenderer);

			// Calculate what to render

			
			size_t blocksX = (SCREEN_WIDTH + BLOCKSIZE - 1) / BLOCKSIZE;
			size_t blocksY = (SCREEN_HEIGHT + BLOCKSIZE - 1) / BLOCKSIZE;

			// Run the propRays kernel
			propRays<<<dim3(blocksX, blocksY), dim3(BLOCKSIZE, BLOCKSIZE)>>>(gpu_frameBuffer, gpu_rgb_image, gpu_rays, dims, gpu_blackHole);

			// Wait for the kernel to finish
			if (hipDeviceSynchronize() != hipSuccess) {
				fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
			}

			// Copy the cpu's rgb_image to the gpu with hipMemcpy
			if (hipMemcpy(frameBuffer, gpu_frameBuffer, sizeof(Rgb) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyDeviceToHost) != hipSuccess) {
				fprintf(stderr, "Failed to copy gpu_frameBuffer back to the CPU\n");
			}

			
			for (int i = 0; i < SCREEN_WIDTH; i++) {
				for (int j = 0; j < SCREEN_HEIGHT; j++) {
					SDL_SetRenderDrawColor(gRenderer, frameBuffer[j * SCREEN_WIDTH + i][0], frameBuffer[j * SCREEN_WIDTH + i][1], frameBuffer[j * SCREEN_WIDTH + i][2], 0xff);
					SDL_RenderDrawPoint(gRenderer, i, j);
				}
			}
			

			// Draw picture to screen
			SDL_RenderPresent(gRenderer);
			
		}
		stbi_image_free(rgb_image);

	}

	
	close();
    
    return 0;
}


void init_rays() {
	printf("Initializing Rays\n");
	for (int i = 0; i < SCREEN_WIDTH; i++) {
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			rays[j * SCREEN_WIDTH + i].pos = cameraPos;
			float theta = (fov * ((float)i / SCREEN_WIDTH) - fov / 2) * (2 * M_PI / 360);
			float phi = (fov * ((float)j / SCREEN_HEIGHT) - fov / 2) * (2 * M_PI / 360);
			rays[j * SCREEN_WIDTH + i].dir = cameraDir;
			rodriguesFormula(&rays[j * SCREEN_WIDTH + i].dir, rays[j * SCREEN_WIDTH + i].dir, cameraUp, theta);
			rodriguesFormula(&rays[j * SCREEN_WIDTH + i].dir, rays[j * SCREEN_WIDTH + i].dir, cameraRight, phi);
			rays[j * SCREEN_WIDTH + i].colored = false;
			//printf("Initializing Ray at Theta: %lf, Phi: %lf\n", theta, phi);
		}
	}
	printf("Rays initialized\n");
}

__global__ void propRays(Rgb* gpu_frameBuffer, uint8_t* gpu_rgb_image, Ray* gpu_rays, Dims* dims, Vec3f_t* gpu_blackHole) {
	int i = threadIdx.x + blockIdx.x * BLOCKSIZE;
	int j = threadIdx.y + blockIdx.y * BLOCKSIZE;
	//printf("Thread: (%d, %d), Block: (%d, %d)\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
	if (i > SCREEN_WIDTH || j > SCREEN_HEIGHT) return;
	if (!gpu_rays[j * SCREEN_WIDTH + i].colored){
		gpu_rays[j * SCREEN_WIDTH + i].pos.x += gpu_rays[j * SCREEN_WIDTH + i].dir.x;
		gpu_rays[j * SCREEN_WIDTH + i].pos.y += gpu_rays[j * SCREEN_WIDTH + i].dir.y;
		gpu_rays[j * SCREEN_WIDTH + i].pos.z += gpu_rays[j * SCREEN_WIDTH + i].dir.z;
		//printf("Moving Ray\n");
		
		float lambda;
		if (gpu_rays[j * SCREEN_WIDTH + i].dir.x > 0)
			lambda = atan(gpu_rays[j * SCREEN_WIDTH + i].dir.y / gpu_rays[j * SCREEN_WIDTH + i].dir.x);
		else if (gpu_rays[j * SCREEN_WIDTH + i].dir.x < 0)
			lambda = atan(gpu_rays[j * SCREEN_WIDTH + i].dir.y / gpu_rays[j * SCREEN_WIDTH + i].dir.x) + M_PI;
		else
			lambda = M_PI / 2;
		float phi = acos(gpu_rays[j * SCREEN_WIDTH + i].dir.z / sqrt(pow(gpu_rays[j * SCREEN_WIDTH + i].dir.x, 2) + pow(gpu_rays[j * SCREEN_WIDTH + i].dir.y, 2) + pow(gpu_rays[j * SCREEN_WIDTH + i].dir.z, 2)));
		int x = lambda / (2 * M_PI) * dims->width;
		int y = phi / M_PI * dims->height;
		gpu_rays[j * SCREEN_WIDTH + i].colored = true;
		if (y * dims->width * 3 + x * 3 < dims->width * dims->height * 3) {
			gpu_frameBuffer[j * SCREEN_WIDTH + i][0] = gpu_rgb_image[y * dims->width * 3 + x * 3];
			gpu_frameBuffer[j * SCREEN_WIDTH + i][1] = gpu_rgb_image[y * dims->width * 3  + x * 3 + 1];
			gpu_frameBuffer[j * SCREEN_WIDTH + i][2] = gpu_rgb_image[y * dims->width * 3 + x * 3 + 2];
		}

	}
}

void rodriguesFormula(Vec3f_t* rotVec, Vec3f_t v, Vec3f_t k, float theta) {

	float kvDot = v.x * k.x + v.y + k.y + v.z * k.z;

	rotVec->x = v.x * cos(theta) + (k.y * v.z - k.z * v.y) * sin(theta) + k.x * (kvDot) * (1 - cos(theta));
	rotVec->y = v.y * cos(theta) + (k.z * v.x- k.x * v.z) * sin(theta) + k.y * (kvDot) * (1 - cos(theta));
	rotVec->z = v.z * cos(theta) + (k.x * v.y - k.y * v.x) * sin(theta) + k.z * (kvDot) * (1 - cos(theta));

}


bool init() {
    bool success = true;

	if (SDL_Init(SDL_INIT_VIDEO) < 0) {
		printf("SDL could not initialize! SDL_Error: %s\n", SDL_GetError());
		success = false;
	}
	else {
		//Create window
		gWindow = SDL_CreateWindow("Black Hole Renderer", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
		if (gWindow == NULL)
		{
			printf("Window could not be created! SDL_Error: %s\n", SDL_GetError());
			success = false;
		}
		else
		{
			gRenderer = SDL_CreateRenderer(gWindow, -1, SDL_RENDERER_ACCELERATED);
			if (gRenderer == NULL) {
				printf("Renderer could not be created! SDL Error: %s\n", SDL_GetError());
				success = false;
			}

		}
	}
	return success;
}

void close() {
	SDL_DestroyRenderer(gRenderer);
	SDL_DestroyWindow(gWindow);
	gWindow = NULL;
	gRenderer = NULL;

	SDL_Quit();
}
