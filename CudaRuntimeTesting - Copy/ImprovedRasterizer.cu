#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <stdbool.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"


const int SCREEN_WIDTH = 1000;
const int SCREEN_HEIGHT = 800;
const int BLOCKSIZE = 8;

typedef struct Vec3f {
	float x;
	float y;
	float z;
} Vec3f_t;

typedef struct Ray {
	Vec3f_t pos;
	Vec3f_t dir;
	bool colored;
};

typedef struct dims {
	int width;
	int height;
} Dims;

typedef unsigned char Rgb[3];

SDL_Window* gWindow = NULL;
SDL_Renderer* gRenderer = NULL;

Vec3f_t cameraPos = { 0, 0, 0.5 };
Vec3f_t cameraDir = { 0, -1, 0 };
Vec3f_t cameraUp = { 0, 0, 1 };
Vec3f_t cameraRight = { -1, 0, 0 };
Vec3f_t globalUp = { 0, 0, 1 };

Vec3f_t blackHolePos = { 0, -11, 0 };

Ray rays[SCREEN_HEIGHT * SCREEN_WIDTH];
Rgb* frameBuffer = new Rgb[SCREEN_WIDTH * SCREEN_HEIGHT];
uint8_t* rgb_image;

float starFieldRadius = 80;
float fov = 70;
float pitch = -0.5;
float yaw = -0.15;

int width, height, bpp;


bool init();

void close();
void init_rays();
__global__ void propRays(Rgb* gpu_frameBuffer, uint8_t* gpu_rgb_image, Ray* gpu_rays, Dims* dims, Vec3f_t* gpu_blackHole);
void rodriguesFormula(Vec3f_t* rotVec, Vec3f_t v, Vec3f_t k, float theta);
void crossProduct(Vec3f_t* destination, Vec3f_t* vec1, Vec3f_t* vec2);



int main(int argc, char** argv){

	Rgb* gpu_frameBuffer;
	uint8_t* gpu_rgb_image;
	Ray* gpu_rays;
	Dims* dims;
	Vec3f_t* gpu_blackHole;

	rgb_image = stbi_load("starmap_2020_4k_brighter.png", &width, &height, &bpp, 3);

	// Allocate space for the frameBuffer on the GPU
	if (hipMalloc(&gpu_frameBuffer, sizeof(Rgb)*SCREEN_HEIGHT*SCREEN_WIDTH) != hipSuccess) {
		fprintf(stderr, "Failed to allocate frameBuffer on GPU\n");
		exit(2);
	}

	// Allocate space for the rgb_image on the GPU
	if (hipMalloc(&gpu_rgb_image, sizeof(uint8_t) * width * height * 3) != hipSuccess) {
		fprintf(stderr, "Failed to allocate rgb_image on GPU\n");
		exit(2);
	}

	// Allocate space for the rays on the GPU
	if (hipMalloc(&gpu_rays, sizeof(Ray) * SCREEN_HEIGHT * SCREEN_WIDTH) != hipSuccess) {
		fprintf(stderr, "Failed to allocate rays on GPU\n");
		exit(2);
	}

	// Allocate space for the rays on the GPU
	if (hipMalloc(&dims, sizeof(Dims)) != hipSuccess) {
		fprintf(stderr, "Failed to allocate dims on GPU\n");
		exit(2);
	}

	// Allocate space for the frameBuffer on the GPU
	if (hipMalloc(&gpu_blackHole, sizeof(Vec3f_t)) != hipSuccess) {
		fprintf(stderr, "Failed to allocate blackHole on GPU\n");
		exit(2);
	}

	if (!rgb_image) {
		fprintf(stderr, "Cannot load file image %s\nSTB Reason: %s\n", "starmap_2020_4k_brighter.png", stbi_failure_reason());
		exit(0);
	}

	// Copy the cpu's rgb_image to the gpu with hipMemcpy
	if (hipMemcpy(gpu_rgb_image, rgb_image, sizeof(uint8_t) * width * height * 3, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy rgb_image to the GPU\n");
		exit(2);
	}

	// Copy the cpu's width to the gpu with hipMemcpy
	if (hipMemcpy(&dims->width, &width, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy width to the GPU\n");
		exit(2);
	}

	// Copy the cpu's height to the gpu with hipMemcpy
	if (hipMemcpy(&dims->height, &height, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy height to the GPU\n");
		exit(2);
	}

	// Copy the cpu's height to the gpu with hipMemcpy
	if (hipMemcpy(gpu_blackHole, &blackHolePos, sizeof(Vec3f_t), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Failed to copy blackHole to the GPU\n");
		exit(2);
	}


	if (!init()) {
		printf("Failed to initialize!\n");
	}
	else {
		init_rays();

		// Copy the cpu's rays to the gpu with hipMemcpy
		if (hipMemcpy(gpu_rays, rays, sizeof(Ray) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyHostToDevice) != hipSuccess) {
			fprintf(stderr, "Failed to copy rays to the GPU\n");
		}

		bool quit = false;
		SDL_Event e;
		while (!quit) {
			while (SDL_PollEvent(&e) != 0) {
				if (e.type == SDL_QUIT) {
					quit = true;
				}
				else if (e.type == SDL_KEYDOWN)
				{
					//Select surfaces based on key press
					switch (e.key.keysym.sym)
					{
					case SDLK_UP:
						if(pitch + 0.1 <= 3.14/2)
							pitch += 0.1;
						break;

					case SDLK_DOWN:
						if(pitch - 0.1 >= -3.14/2)
							pitch -= 0.1;
						break;

					case SDLK_LEFT:
						yaw -= 0.1;
						break;

					case SDLK_RIGHT:
						yaw += 0.1;
						break;
					case SDLK_ESCAPE:
						close();
						return 0;
						break;
					}
					printf("Camera Dir: (%lf, %lf, %lf)\n", cameraDir.x, cameraDir.y, cameraDir.z);
					printf("Camera Up: (%lf, %lf, %lf)\n", cameraUp.x, cameraUp.y, cameraUp.z);
					printf("Camera Right: (%lf, %lf, %lf)\n", cameraRight.x, cameraRight.y, cameraRight.z);
					printf("Yaw: %lf\n", yaw);
					printf("Pitch: %lf\n", pitch);

					init_rays();

					// Copy the cpu's rays to the gpu with hipMemcpy
					if (hipMemcpy(gpu_rays, rays, sizeof(Ray) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyHostToDevice) != hipSuccess) {
						fprintf(stderr, "Failed to copy rays to the GPU\n");
					}
				}
			}
			


			SDL_SetRenderDrawColor(gRenderer, 0x00, 0x00, 0x00, 0x00);
			SDL_RenderClear(gRenderer);

			// Calculate what to render

			
			size_t blocksX = (SCREEN_WIDTH + BLOCKSIZE - 1) / BLOCKSIZE;
			size_t blocksY = (SCREEN_HEIGHT + BLOCKSIZE - 1) / BLOCKSIZE;

			// Run the propRays kernel
			propRays<<<dim3(blocksX, blocksY), dim3(BLOCKSIZE, BLOCKSIZE)>>>(gpu_frameBuffer, gpu_rgb_image, gpu_rays, dims, gpu_blackHole);

			// Wait for the kernel to finish
			if (hipDeviceSynchronize() != hipSuccess) {
				fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
			}

			// Copy the cpu's rgb_image to the gpu with hipMemcpy
			if (hipMemcpy(frameBuffer, gpu_frameBuffer, sizeof(Rgb) * SCREEN_HEIGHT * SCREEN_WIDTH, hipMemcpyDeviceToHost) != hipSuccess) {
				fprintf(stderr, "Failed to copy gpu_frameBuffer back to the CPU\n");
			}

			
			for (int i = 0; i < SCREEN_WIDTH; i++) {
				for (int j = 0; j < SCREEN_HEIGHT; j++) {
					SDL_SetRenderDrawColor(gRenderer, frameBuffer[j * SCREEN_WIDTH + i][0], frameBuffer[j * SCREEN_WIDTH + i][1], frameBuffer[j * SCREEN_WIDTH + i][2], 0xff);
					SDL_RenderDrawPoint(gRenderer, i, j);
				}
			}
			

			// Draw picture to screen
			SDL_RenderPresent(gRenderer);
			
		}
		stbi_image_free(rgb_image);

	}

	
	close();
    
    return 0;
}


void init_rays() {
	printf("Initializing Rays\n");
	cameraDir = { 0, -1, 0 };
	cameraUp = { 0, 0, 1 };
	cameraRight = { -1, 0, 0 };
	rodriguesFormula(&cameraDir, cameraDir, cameraRight, pitch);
	rodriguesFormula(&cameraUp, cameraUp, cameraRight, pitch);
	rodriguesFormula(&cameraDir, cameraDir, globalUp, yaw);
	rodriguesFormula(&cameraRight, cameraRight, globalUp, yaw);
	for (int i = 0; i < SCREEN_WIDTH; i++) {
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			rays[j * SCREEN_WIDTH + i].pos = cameraPos;
			Vec3f_t toPos;
			float aspectRatio = (float)SCREEN_WIDTH / SCREEN_HEIGHT;
			toPos.x = cameraPos.x + cameraDir.x * 0.8 + (-((float)i / SCREEN_WIDTH) + 0.5) * cameraRight.x * aspectRatio + (-((float)j / SCREEN_HEIGHT) + 0.5) * cameraUp.x;
			toPos.y = cameraPos.y + cameraDir.y * 0.8 + (-((float)i / SCREEN_WIDTH) + 0.5) * cameraRight.y * aspectRatio + (-((float)j / SCREEN_HEIGHT) + 0.5) * cameraUp.y;
			toPos.z = cameraPos.z + cameraDir.z * 0.8 + (-((float)i / SCREEN_WIDTH) + 0.5) * cameraRight.z * aspectRatio + (-((float)j / SCREEN_HEIGHT) + 0.5) * cameraUp.z;

			float mag = sqrt(pow(toPos.x, 2) + pow(toPos.y, 2) + pow(toPos.z, 2));

			toPos.x = toPos.x / mag;
			toPos.y = toPos.y / mag;
			toPos.z = toPos.z / mag;
			//printf("Ray at: (%.2lf, %.2lf)\n", (fov * ((float)i / SCREEN_WIDTH) - fov / 2), (fov * (-(float)j / SCREEN_HEIGHT) + fov / 2));
			//float theta = (fov * ((float)i / SCREEN_WIDTH) - fov / 2) * (2 * M_PI / 360);
			//float phi = (fov * (-(float)j / SCREEN_HEIGHT) + fov / 2) * (2 * M_PI / 360);
			rays[j * SCREEN_WIDTH + i].dir = toPos;
			//rodriguesFormula(&rays[j * SCREEN_WIDTH + i].dir, rays[j * SCREEN_WIDTH + i].dir, cameraUp, theta);
			//rodriguesFormula(&rays[j * SCREEN_WIDTH + i].dir, rays[j * SCREEN_WIDTH + i].dir, cameraRight, phi);
			rays[j * SCREEN_WIDTH + i].colored = false;
			//printf("Initializing Ray at Theta: %lf, Phi: %lf\n", theta, phi);
		}
	}
	printf("Rays initialized\n");
}

__global__ void propRays(Rgb* gpu_frameBuffer, uint8_t* gpu_rgb_image, Ray* gpu_rays, Dims* dims, Vec3f_t* gpu_blackHole) {
	float stepSize = 0.01;
	float bhRadius = 3;
	int rings = 10;
	int i = threadIdx.x + blockIdx.x * BLOCKSIZE;
	int j = threadIdx.y + blockIdx.y * BLOCKSIZE;
	//printf("Thread: (%d, %d), Block: (%d, %d)\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
	if (i > SCREEN_WIDTH || j > SCREEN_HEIGHT) return;
	if (!gpu_rays[j * SCREEN_WIDTH + i].colored){

		Vec3f_t toBHOld;
		toBHOld.x = gpu_blackHole->x - gpu_rays[j * SCREEN_WIDTH + i].pos.x;
		toBHOld.y = gpu_blackHole->y - gpu_rays[j * SCREEN_WIDTH + i].pos.y;
		toBHOld.z = gpu_blackHole->z - gpu_rays[j * SCREEN_WIDTH + i].pos.z;

		float rOld = sqrt(pow(toBHOld.x, 2) + pow(toBHOld.y, 2) + pow(toBHOld.z, 2));

		float rho = pow(gpu_rays[j * SCREEN_WIDTH + i].dir.x, 2) + pow(gpu_rays[j * SCREEN_WIDTH + i].dir.y, 2) + pow(gpu_rays[j * SCREEN_WIDTH + i].dir.z, 2);

		float ax = -gpu_rays[j * SCREEN_WIDTH + i].dir.x * gpu_rays[j * SCREEN_WIDTH + i].pos.x + gpu_rays[j * SCREEN_WIDTH + i].dir.x * gpu_blackHole->x;
		float ay = -gpu_rays[j * SCREEN_WIDTH + i].dir.y * gpu_rays[j * SCREEN_WIDTH + i].pos.y + gpu_rays[j * SCREEN_WIDTH + i].dir.y * gpu_blackHole->y;
		float az = -gpu_rays[j * SCREEN_WIDTH + i].dir.z * gpu_rays[j * SCREEN_WIDTH + i].pos.z + gpu_rays[j * SCREEN_WIDTH + i].dir.z * gpu_blackHole->z;
		float a = (ax + ay + az) / rho;

		Vec3f_t perihelion;
		perihelion.x = gpu_rays[j * SCREEN_WIDTH + i].pos.x + a * gpu_rays[j * SCREEN_WIDTH + i].dir.x;
		perihelion.y = gpu_rays[j * SCREEN_WIDTH + i].pos.y + a * gpu_rays[j * SCREEN_WIDTH + i].dir.y;
		perihelion.z = gpu_rays[j * SCREEN_WIDTH + i].pos.z + a * gpu_rays[j * SCREEN_WIDTH + i].dir.z;

		float b = sqrt(pow(gpu_blackHole->x - perihelion.x, 2) + pow(gpu_blackHole->y - perihelion.y, 2) + pow(gpu_blackHole->z - perihelion.z, 2));

		Vec3f_t radiusVec;
		radiusVec.x = gpu_blackHole->x - perihelion.x;
		radiusVec.y = gpu_blackHole->y - perihelion.y;
		radiusVec.z = gpu_blackHole->z - perihelion.z;

		Vec3f_t normal;
		normal.x = gpu_rays[j * SCREEN_WIDTH + i].dir.y * radiusVec.z - gpu_rays[j * SCREEN_WIDTH + i].dir.z * radiusVec.y;
		normal.y = gpu_rays[j * SCREEN_WIDTH + i].dir.z * radiusVec.x - gpu_rays[j * SCREEN_WIDTH + i].dir.x * radiusVec.z;
		normal.z = gpu_rays[j * SCREEN_WIDTH + i].dir.x * radiusVec.y - gpu_rays[j * SCREEN_WIDTH + i].dir.y * radiusVec.x;

		gpu_rays[j * SCREEN_WIDTH + i].pos.x += gpu_rays[j * SCREEN_WIDTH + i].dir.x * stepSize;
		gpu_rays[j * SCREEN_WIDTH + i].pos.y += gpu_rays[j * SCREEN_WIDTH + i].dir.y * stepSize;
		gpu_rays[j * SCREEN_WIDTH + i].pos.z += gpu_rays[j * SCREEN_WIDTH + i].dir.z * stepSize;

		Vec3f_t toBH;
		toBH.x = gpu_blackHole->x - gpu_rays[j * SCREEN_WIDTH + i].pos.x;
		toBH.y = gpu_blackHole->y - gpu_rays[j * SCREEN_WIDTH + i].pos.y;
		toBH.z = gpu_blackHole->z - gpu_rays[j * SCREEN_WIDTH + i].pos.z;

		float r = sqrt(pow(toBH.x, 2) + pow(toBH.y, 2) + pow(toBH.z, 2));

		float dr = r - rOld;

		float theta = dr / (pow(r, 2) * sqrt((1 / pow(b, 2)) - (1 - bhRadius / r) * (1 / pow(r, 2))));

		Vec3f_t rotVec;
		Vec3f_t v = gpu_rays[j * SCREEN_WIDTH + i].dir;
		Vec3f_t k = normal;
		float kvDot = gpu_rays[j * SCREEN_WIDTH + i].dir.x * normal.x +
			gpu_rays[j * SCREEN_WIDTH + i].dir.y * normal.y +
			gpu_rays[j * SCREEN_WIDTH + i].dir.z * normal.z;

		rotVec.x = v.x * cos(theta) + (k.y * v.z - k.z * v.y) * sin(theta) + k.x * (kvDot) * (1 - cos(theta));
		rotVec.y = v.y * cos(theta) + (k.z * v.x - k.x * v.z) * sin(theta) + k.y * (kvDot) * (1 - cos(theta));
		rotVec.z = v.z * cos(theta) + (k.x * v.y - k.y * v.x) * sin(theta) + k.z * (kvDot) * (1 - cos(theta));

		gpu_rays[j * SCREEN_WIDTH + i].dir = rotVec;
		//printf("Dist to BH: %lf\n", distToBH);

		
		if (r <= bhRadius) {
			gpu_frameBuffer[j * SCREEN_WIDTH + i][0] = 0;
			gpu_frameBuffer[j * SCREEN_WIDTH + i][1] = 0;
			gpu_frameBuffer[j * SCREEN_WIDTH + i][2] = 0;
			gpu_rays[j * SCREEN_WIDTH + i].colored = true;
			//printf("Blackhole hit\n");
			return;
		}
		else if (r < bhRadius + 3) {
			float angle = acos(gpu_rays[j * SCREEN_WIDTH + i].pos.z / r);
			if (angle < M_PI / 2 + 0.005 && angle > M_PI / 2 - 0.005) {
				gpu_frameBuffer[j * SCREEN_WIDTH + i][0] = r * 255 * rings;
				gpu_frameBuffer[j * SCREEN_WIDTH + i][1] = r * 255 * rings;
				gpu_frameBuffer[j * SCREEN_WIDTH + i][2] = r * 255 * rings;
				gpu_rays[j * SCREEN_WIDTH + i].colored = true;
				return;
				//printf("Acretion Disk hit\n");
			}

		}
		

		//printf("Moving Ray\n");
		if (!gpu_rays[j * SCREEN_WIDTH + i].colored && r >= 30) {

			/*
			gpu_frameBuffer[j * SCREEN_WIDTH + i][0] = 40;
			gpu_frameBuffer[j * SCREEN_WIDTH + i][1] = 40;
			gpu_frameBuffer[j * SCREEN_WIDTH + i][2] = 40;
			gpu_rays[j * SCREEN_WIDTH + i].colored = true;
			return;
			*/

			//printf("Dist to BH: %lf\n", distToBH);
			
			float rho = pow(gpu_rays[j * SCREEN_WIDTH + i].dir.x, 2) + pow(gpu_rays[j * SCREEN_WIDTH + i].dir.y, 2) + pow(gpu_rays[j * SCREEN_WIDTH + i].dir.z, 2);
			gpu_rays[j * SCREEN_WIDTH + i].dir.x = gpu_rays[j * SCREEN_WIDTH + i].dir.x / sqrt(rho);
			gpu_rays[j * SCREEN_WIDTH + i].dir.y = gpu_rays[j * SCREEN_WIDTH + i].dir.y / sqrt(rho);
			gpu_rays[j * SCREEN_WIDTH + i].dir.z = gpu_rays[j * SCREEN_WIDTH + i].dir.z / sqrt(rho);
			
			float lambda;
			if (gpu_rays[j * SCREEN_WIDTH + i].dir.x > 0)
				lambda = atan(gpu_rays[j * SCREEN_WIDTH + i].dir.y / gpu_rays[j * SCREEN_WIDTH + i].dir.x);
			else if (gpu_rays[j * SCREEN_WIDTH + i].dir.x < 0)
				lambda = atan(gpu_rays[j * SCREEN_WIDTH + i].dir.y / gpu_rays[j * SCREEN_WIDTH + i].dir.x) + M_PI;
			else
				lambda = M_PI / 2;
			float phi = acos(gpu_rays[j * SCREEN_WIDTH + i].dir.z);
			int x = ((lambda) / (2 * M_PI)) * dims->width;
			int y = ((phi) / M_PI) * dims->height;
			//printf("Drawing (%d, %d)\n", x, y);
			gpu_rays[j * SCREEN_WIDTH + i].colored = true;
			if (y * dims->width * 3 + x * 3 < dims->width * dims->height * 3) {
				gpu_frameBuffer[j * SCREEN_WIDTH + i][0] = gpu_rgb_image[y * dims->width * 3 + x * 3];
				gpu_frameBuffer[j * SCREEN_WIDTH + i][1] = gpu_rgb_image[y * dims->width * 3 + x * 3 + 1];
				gpu_frameBuffer[j * SCREEN_WIDTH + i][2] = gpu_rgb_image[y * dims->width * 3 + x * 3 + 2];
			}
			
		}
	}
}

void rodriguesFormula(Vec3f_t* rotVec, Vec3f_t v, Vec3f_t k, float theta) {

	float kvDot = v.x * k.x + v.y + k.y + v.z * k.z;

	Vec3f_t newVec;

	newVec.x = v.x * cos(theta) + (k.y * v.z - k.z * v.y) * sin(theta) + k.x * (kvDot) * (1 - cos(theta));
	newVec.y = v.y * cos(theta) + (k.z * v.x - k.x * v.z) * sin(theta) + k.y * (kvDot) * (1 - cos(theta));
	newVec.z = v.z * cos(theta) + (k.x * v.y - k.y * v.x) * sin(theta) + k.z * (kvDot) * (1 - cos(theta));
	*rotVec = newVec;
}

void crossProduct(Vec3f_t* destination, Vec3f_t* vec1, Vec3f_t* vec2) {
	destination->x = vec1->y * vec2->z - vec1->z * vec2->y;
	destination->y = vec1->z * vec2->x - vec1->x * vec2->z;
	destination->z = vec1->x * vec2->y - vec1->y * vec2->x;
}


bool init() {
    bool success = true;

	if (SDL_Init(SDL_INIT_VIDEO) < 0) {
		printf("SDL could not initialize! SDL_Error: %s\n", SDL_GetError());
		success = false;
	}
	else {
		//Create window
		gWindow = SDL_CreateWindow("Black Hole Renderer", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
		if (gWindow == NULL)
		{
			printf("Window could not be created! SDL_Error: %s\n", SDL_GetError());
			success = false;
		}
		else
		{
			gRenderer = SDL_CreateRenderer(gWindow, -1, SDL_RENDERER_ACCELERATED);
			if (gRenderer == NULL) {
				printf("Renderer could not be created! SDL Error: %s\n", SDL_GetError());
				success = false;
			}

		}
	}
	return success;
}

void close() {
	SDL_DestroyRenderer(gRenderer);
	SDL_DestroyWindow(gWindow);
	gWindow = NULL;
	gRenderer = NULL;

	SDL_Quit();
}
