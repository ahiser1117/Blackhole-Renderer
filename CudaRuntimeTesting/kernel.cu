#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <stdbool.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>


const int SCREEN_WIDTH = 512;
const int SCREEN_HEIGHT = 512;

typedef struct Vec3f {
	float x;
	float y;
	float z;
} Vec3f_t;

typedef struct Poly {
	Vec3f_t vertices[3];
	char rgb[3];
} Poly_t;

typedef struct Object {
	Poly_t* tris;
	size_t size;
} Object_t;


SDL_Window* gWindow = NULL;
SDL_Renderer* gRenderer = NULL;

Vec3f_t* cameraPos;
Vec3f_t* cameraDir;

float near = 1;
float scale = 1;

bool init();

void close();

void renderObject(Object_t* object);
void rotateZAxis(Object_t* object, float theta);
float edgeFunction(const Vec3f_t& a, const Vec3f_t& b, const Vec3f_t& c);


int main(int argc, char** argv)
{

	Object_t* object1 = (Object_t*)malloc(sizeof(Object_t));

	object1->size = 6;
	object1->tris = (Poly_t*)malloc(sizeof(Poly_t) * object1->size);


	// Pos y face
	object1->tris[0].vertices[0] = { 1, 1, 7 };
	object1->tris[0].vertices[1] = { -1, 1, 7 };
	object1->tris[0].vertices[2] = { 1, 1, 5 };
	object1->tris[0].rgb[0] = 255;
	object1->tris[0].rgb[1] = 0;
	object1->tris[0].rgb[2] = 0;

	object1->tris[1].vertices[0] = { -1, 1, 5 };
	object1->tris[1].vertices[1] = { -1, 1, 7 };
	object1->tris[1].vertices[2] = { 1, 1, 5 };
	object1->tris[1].rgb[0] = 0;
	object1->tris[1].rgb[1] = 255;
	object1->tris[1].rgb[2] = 0;

	// Neg z face
	object1->tris[2].vertices[0] = { 1, 1, 5 };
	object1->tris[2].vertices[1] = { 1, -1, 5 };
	object1->tris[2].vertices[2] = { -1, -1, 5 };
	object1->tris[2].rgb[0] = 255;
	object1->tris[2].rgb[1] = 0;
	object1->tris[2].rgb[2] = 0;

	object1->tris[3].vertices[0] = { 1, 1, 5 };
	object1->tris[3].vertices[1] = { -1, 1, 5 };
	object1->tris[3].vertices[2] = { -1, -1, 5 };
	object1->tris[3].rgb[0] = 0;
	object1->tris[3].rgb[1] = 0;
	object1->tris[3].rgb[2] = 0;

	// Neg y face
	
	object1->tris[4].vertices[0] = { 1, -1, 7 };
	object1->tris[4].vertices[1] = { 1, -1, 5 };
	object1->tris[4].vertices[2] = { -1, -1, 5 };
	object1->tris[4].rgb[0] = 255;
	object1->tris[4].rgb[1] = 0;
	object1->tris[4].rgb[2] = 0;

	object1->tris[5].vertices[0] = { 1, -1, 7 };
	object1->tris[5].vertices[1] = { -1, -1, 7 };
	object1->tris[5].vertices[2] = { -1, -1, 5 };
	object1->tris[5].rgb[0] = 0;
	object1->tris[5].rgb[1] = 0;
	object1->tris[5].rgb[2] = 255;
	


	if (!init()) {
		printf("Failed to initialize!\n");
	}
	else {

		bool quit = false;
		SDL_Event e;
		while (!quit) {
			while (SDL_PollEvent(&e) != 0) {
				if (e.type == SDL_QUIT) {
					quit = true;
				}
			}
			/*
			SDL_SetRenderDrawColor(gRenderer, 0xff, 0xff, 0xff, 0xff);
			SDL_RenderClear(gRenderer);

			SDL_Rect fillRect = { SCREEN_WIDTH / 4, SCREEN_HEIGHT / 4, SCREEN_WIDTH / 2, SCREEN_HEIGHT / 2 };
			SDL_SetRenderDrawColor(gRenderer, 0xff, 0x00, 0x00, 0xff);
			SDL_RenderFillRect(gRenderer, &fillRect);

			SDL_Rect outlineRect = { SCREEN_WIDTH / 6, SCREEN_HEIGHT / 6, SCREEN_WIDTH * 2 / 3, SCREEN_HEIGHT * 2 / 3 };
			SDL_SetRenderDrawColor(gRenderer, 0x00, 0xff, 0x00, 0xff);
			SDL_RenderDrawRect(gRenderer, &outlineRect);

			SDL_SetRenderDrawColor(gRenderer, 0x00, 0x00, 0xff, 0xff);
			SDL_RenderDrawLine(gRenderer, 0, SCREEN_HEIGHT / 2, SCREEN_WIDTH, SCREEN_HEIGHT / 2);

			SDL_SetRenderDrawColor(gRenderer, 0xff, 0xff, 0x00, 0xff);
			for (int i = 0; i < SCREEN_HEIGHT; i += 4) {
				SDL_RenderDrawPoint(gRenderer, SCREEN_WIDTH / 2, i);
			}
			*/
			SDL_SetRenderDrawColor(gRenderer, 0x00, 0x00, 0x00, 0x00);
			SDL_RenderClear(gRenderer);

			renderObject(object1);
			rotateZAxis(object1, 0.01);
			



			SDL_RenderPresent(gRenderer);


		}

	}

	close();
    
    return 0;
}

float edgeFunction(const Vec3f_t& a, const Vec3f_t& b, const Vec3f_t& c) {
	return (c.x - a.x) * (b.y - a.y) - (c.y - a.y) * (b.x - a.x);
}

void renderObject(Object_t* object) {
	Poly_t* screenSpace = (Poly_t*)malloc(sizeof(Poly_t) * object->size);

	float t, b, r, l;

	// Convert to Screen Space
	for (size_t i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			screenSpace[i].vertices[j].x =
				(near * object->tris[i].vertices[j].x) /
				(-object->tris[i].vertices[j].z);
			screenSpace[i].vertices[j].y =
				(near * object->tris[i].vertices[j].y) /
				(-object->tris[i].vertices[j].z);
		}
	}

	t = 1 / scale;
	b = -t;
	r = 1 / scale;
	l = -r;

	Poly_t* NDCSpace = (Poly_t*)malloc(sizeof(Poly_t) * object->size);

	// Convert to NDC Space
	for (size_t i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			NDCSpace[i].vertices[j].x =
				(2 * screenSpace[i].vertices[j].x) / (r - l) - (r + l) / (r - l);
			NDCSpace[i].vertices[j].y =
				(2 * screenSpace[i].vertices[j].y) / (t - b) - (t + b) / (t - b);
		}
	}

	Poly_t* rasterSpace = (Poly_t*)malloc(sizeof(Poly_t) * object->size);

	for (size_t i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			rasterSpace[i].vertices[j].x =
				(NDCSpace[i].vertices[j].x + 1) / 2 * SCREEN_WIDTH;
			rasterSpace[i].vertices[j].y =
				(1 - NDCSpace[i].vertices[j].y) / 2 * SCREEN_HEIGHT;
			rasterSpace[i].vertices[j].z = -object->tris[i].vertices[j].z;
			SDL_SetRenderDrawColor(gRenderer, 0xff, 0x0, 0x0, 0xff);
			SDL_RenderDrawPoint(gRenderer, rasterSpace[i].vertices[j].x, rasterSpace[i].vertices[j].y);
		}
	}
	for (int k = 0; k < object->size; k++) {
		float area = edgeFunction(rasterSpace[k].vertices[0], rasterSpace[k].vertices[1], rasterSpace[k].vertices[2]);
		for (int j = 0; j < SCREEN_HEIGHT; j++) {
			for (int i = 0; i < SCREEN_WIDTH; i++) {
				Vec3f_t p = { i, j, 0 };
				float w0 = edgeFunction(rasterSpace[k].vertices[1], rasterSpace[k].vertices[2], p);
				float w1 = edgeFunction(rasterSpace[k].vertices[2], rasterSpace[k].vertices[0], p);
				float w2 = edgeFunction(rasterSpace[k].vertices[0], rasterSpace[k].vertices[1], p);
				if (w0 >= 0 && w1 >= 0 && w2 >= 0) {
					//printf("Draw Point with color: %d, %d, %d\n", rasterS[k].rgb[0], rasterSpace[k].rgb[1], rasterSpace[k].rgb[2]);
					SDL_SetRenderDrawColor(gRenderer, object->tris[k].rgb[0], object->tris[k].rgb[1], object->tris[k].rgb[2], 0xff);
					//printf("Drawing point at (%lf, %lf)\n", rasterSpace[i].vertices[j].x, rasterSpace[i].vertices[j].y);
					SDL_RenderDrawPoint(gRenderer, p.x, p.y);
				}
			}
		}
	}
	


}

void rotateZAxis(Object_t* object, float theta) {

	for (int i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			Vec3f_t rotated;
			rotated.x = object->tris[i].vertices[j].x * cos(theta) - object->tris[i].vertices[j].y * sin(theta);
			rotated.y = object->tris[i].vertices[j].y * cos(theta) + object->tris[i].vertices[j].x * sin(theta);
			object->tris[i].vertices[j].x = rotated.x;
			object->tris[i].vertices[j].y = rotated.y;
		}
	}


}


bool init() {
    bool success = true;

	if (SDL_Init(SDL_INIT_VIDEO) < 0) {
		printf("SDL could not initialize! SDL_Error: %s\n", SDL_GetError());
		success = false;
	}
	else {
		//Create window
		gWindow = SDL_CreateWindow("SDL Tutorial", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
		if (gWindow == NULL)
		{
			printf("Window could not be created! SDL_Error: %s\n", SDL_GetError());
			success = false;
		}
		else
		{
			gRenderer = SDL_CreateRenderer(gWindow, -1, SDL_RENDERER_ACCELERATED);
			if (gRenderer == NULL) {
				printf("Renderer could not be created! SDL Error: %s\n", SDL_GetError());
				success = false;
			}

		}
	}
	return success;
}

void close() {
	SDL_DestroyRenderer(gRenderer);
	SDL_DestroyWindow(gWindow);
	gWindow = NULL;
	gRenderer = NULL;

	SDL_Quit();
}
