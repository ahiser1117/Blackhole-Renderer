#include "hip/hip_runtime.h"
﻿#include <SDL.h>
#include <stdbool.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>


const int SCREEN_WIDTH = 512;
const int SCREEN_HEIGHT = 512;

typedef struct Vec3f {
	float x;
	float y;
	float z;
} Vec3f_t;

typedef struct RBG {
	short r;
	short g;
	short b;
} RGB;

typedef struct Poly {
	Vec3f_t vertices[3];
	RGB rgb;
} Poly_t;

typedef struct Object {
	Poly_t* tris;
	size_t size;
	Vec3f_t pos;
} Object_t;


SDL_Window* gWindow = NULL;
SDL_Renderer* gRenderer = NULL;

Vec3f_t* cameraPos;
Vec3f_t* cameraDir;
float zBuffer[SCREEN_HEIGHT * SCREEN_WIDTH] = {INFINITY};

float near = 1;
float scale = 1;

bool init();

void close();

void renderObject(Object_t* object);
void rotateZAxis(Object_t* object, float theta);
void rotateXAxis(Object_t* object, float theta);
void rotateYAxis(Object_t* object, float theta);
float edgeFunction(const Vec3f_t& a, const Vec3f_t& b, const Vec3f_t& c);


int main(int argc, char** argv)
{

	Object_t* object1 = (Object_t*)malloc(sizeof(Object_t));

	object1->size = 12;
	object1->tris = (Poly_t*)malloc(sizeof(Poly_t) * object1->size);
	object1->pos = { 0, 0, 5 };

	// Pos y face
	object1->tris[0].vertices[0] = { 1, 1, 1 };
	object1->tris[0].vertices[1] = { -1, 1, 1 };
	object1->tris[0].vertices[2] = { 1, 1, -1 };
	object1->tris[0].rgb = { 0, 0, 255 };

	object1->tris[1].vertices[1] = { -1, 1, -1 };
	object1->tris[1].vertices[0] = { -1, 1, 1 };
	object1->tris[1].vertices[2] = { 1, 1, -1 };
	object1->tris[1].rgb = { 0,255,0 };

	// Neg z face
	object1->tris[2].vertices[1] = { 1, 1, -1 };
	object1->tris[2].vertices[0] = { 1, -1, -1 };
	object1->tris[2].vertices[2] = { -1, -1, -1 };
	object1->tris[2].rgb = { 255, 0,0 };

	object1->tris[3].vertices[0] = { 1, 1, -1 };
	object1->tris[3].vertices[1] = { -1, 1, -1 };
	object1->tris[3].vertices[2] = { -1, -1, -1 };
	object1->tris[3].rgb = { 0,0,255 };

	// Neg y face
	
	object1->tris[4].vertices[0] = { 1, -1, 1 };
	object1->tris[4].vertices[1] = { 1, -1, -1 };
	object1->tris[4].vertices[2] = { -1, -1, -1 };
	object1->tris[4].rgb = { 0,255,0 };

	object1->tris[5].vertices[1] = { 1, -1, 1 };
	object1->tris[5].vertices[0] = { -1, -1, 1 };
	object1->tris[5].vertices[2] = { -1, -1, -1 };
	object1->tris[5].rgb = { 255,255,255 };

	// Pos Z face

	object1->tris[6].vertices[1] = { 1, 1, 1 };
	object1->tris[6].vertices[0] = { -1, 1, 1 };
	object1->tris[6].vertices[2] = { -1, -1, 1 };
	object1->tris[6].rgb = { 0,255,0 };

	object1->tris[7].vertices[1] = { 1, 1, 1 };
	object1->tris[7].vertices[0] = { -1, -1, 1 };
	object1->tris[7].vertices[2] = { 1, -1, 1 };
	object1->tris[7].rgb = { 0,0,255 };

	// Pos X face

	object1->tris[8].vertices[1] = { 1, -1, -1 };
	object1->tris[8].vertices[0] = { 1, 1, -1 };
	object1->tris[8].vertices[2] = { 1, 1, 1 };
	object1->tris[8].rgb = { 0,255,0 };

	object1->tris[9].vertices[1] = { 1, -1, -1 };
	object1->tris[9].vertices[0] = { 1, 1, 1 };
	object1->tris[9].vertices[2] = { 1, -1, 1 };
	object1->tris[9].rgb = { 0,0,255 };

	// Neg X face

	object1->tris[10].vertices[0] = { -1, -1, -1 };
	object1->tris[10].vertices[1] = { -1, 1, -1 };
	object1->tris[10].vertices[2] = { -1, 1, 1 };
	object1->tris[10].rgb = { 0,255,0 };

	object1->tris[11].vertices[0] = { -1, -1, -1 };
	object1->tris[11].vertices[1] = { -1, 1, 1 };
	object1->tris[11].vertices[2] = { -1, -1, 1 };
	object1->tris[11].rgb = { 0,0,255 };
	


	if (!init()) {
		printf("Failed to initialize!\n");
	}
	else {

		bool quit = false;
		SDL_Event e;
		while (!quit) {
			while (SDL_PollEvent(&e) != 0) {
				if (e.type == SDL_QUIT) {
					quit = true;
				}
			}
			/*
			SDL_SetRenderDrawColor(gRenderer, 0xff, 0xff, 0xff, 0xff);
			SDL_RenderClear(gRenderer);

			SDL_Rect fillRect = { SCREEN_WIDTH / 4, SCREEN_HEIGHT / 4, SCREEN_WIDTH / 2, SCREEN_HEIGHT / 2 };
			SDL_SetRenderDrawColor(gRenderer, 0xff, 0x00, 0x00, 0xff);
			SDL_RenderFillRect(gRenderer, &fillRect);

			SDL_Rect outlineRect = { SCREEN_WIDTH / 6, SCREEN_HEIGHT / 6, SCREEN_WIDTH * 2 / 3, SCREEN_HEIGHT * 2 / 3 };
			SDL_SetRenderDrawColor(gRenderer, 0x00, 0xff, 0x00, 0xff);
			SDL_RenderDrawRect(gRenderer, &outlineRect);

			SDL_SetRenderDrawColor(gRenderer, 0x00, 0x00, 0xff, 0xff);
			SDL_RenderDrawLine(gRenderer, 0, SCREEN_HEIGHT / 2, SCREEN_WIDTH, SCREEN_HEIGHT / 2);

			SDL_SetRenderDrawColor(gRenderer, 0xff, 0xff, 0x00, 0xff);
			for (int i = 0; i < SCREEN_HEIGHT; i += 4) {
				SDL_RenderDrawPoint(gRenderer, SCREEN_WIDTH / 2, i);
			}
			*/
			SDL_SetRenderDrawColor(gRenderer, 0x00, 0x00, 0x00, 0x00);
			SDL_RenderClear(gRenderer);

			renderObject(object1);
			//rotateXAxis(object1, 0.07);
			//rotateZAxis(object1, 0.05);
			rotateYAxis(object1, 0.03);

			SDL_RenderPresent(gRenderer);
		}

	}

	close();
    
    return 0;
}

float edgeFunction(const Vec3f_t& a, const Vec3f_t& b, const Vec3f_t& c) {
	return (c.x - a.x) * (b.y - a.y) - (c.y - a.y) * (b.x - a.x);
}

void renderObject(Object_t* object) {
	Poly_t* screenSpace = (Poly_t*)malloc(sizeof(Poly_t) * object->size);

	float t, b, r, l;

	// Convert to Screen Space
	for (size_t i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			screenSpace[i].vertices[j].x =
				(near * (object->tris[i].vertices[j].x + object->pos.x)) /
				(-object->tris[i].vertices[j].z - object->pos.z);
			screenSpace[i].vertices[j].y =
				(near * (object->tris[i].vertices[j].y + object->pos.y)) /
				(-object->tris[i].vertices[j].z - object->pos.z);
		}
	}

	t = 1 / scale;
	b = -t;
	r = 1 / scale;
	l = -r;

	Poly_t* NDCSpace = (Poly_t*)malloc(sizeof(Poly_t) * object->size);

	// Convert to NDC Space
	for (size_t i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			NDCSpace[i].vertices[j].x =
				(2 * screenSpace[i].vertices[j].x) / (r - l) - (r + l) / (r - l);
			NDCSpace[i].vertices[j].y =
				(2 * screenSpace[i].vertices[j].y) / (t - b) - (t + b) / (t - b);
		}
	}

	Poly_t* rasterSpace = (Poly_t*)malloc(sizeof(Poly_t) * object->size);

	for (size_t i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			rasterSpace[i].vertices[j].x =
				(NDCSpace[i].vertices[j].x + 1) / 2 * SCREEN_WIDTH;
			rasterSpace[i].vertices[j].y =
				(1 - NDCSpace[i].vertices[j].y) / 2 * SCREEN_HEIGHT;
			rasterSpace[i].vertices[j].z = -object->tris[i].vertices[j].z - object->pos.z;
		}
	}

	// Loop over all triangles in the object
	for (int k = 0; k < object->size; k++) {
		float area = edgeFunction(rasterSpace[k].vertices[0], rasterSpace[k].vertices[1], rasterSpace[k].vertices[2]);
		// Calculate Bounding Box for triangle
		Vec3f_t bbmin, bbmax;
		bbmin.x = INFINITY;
		bbmin.y = INFINITY;
		bbmax.x = -INFINITY;
		bbmax.y = -INFINITY;
		for (int p = 0; p < 3; p++) {
			Vec3f_t vertex = rasterSpace[k].vertices[p];
			if (vertex.x < bbmin.x) bbmin.x = vertex.x;
			if (vertex.y < bbmin.y) bbmin.y = vertex.y;
			if (vertex.x > bbmax.x) bbmax.x = vertex.x;
			if (vertex.y > bbmax.y) bbmax.y = vertex.y;
		}
		// Check if each pixel in bounding box is on triangle
		for (int j = (int)bbmin.y; j < (int)bbmax.y; j++) {
			for (int i = (int)bbmin.x; i < (int)bbmax.x; i++) {
				Vec3f_t p = { i, j, 0 };
				float w0 = edgeFunction(rasterSpace[k].vertices[1], rasterSpace[k].vertices[2], p);
				float w1 = edgeFunction(rasterSpace[k].vertices[2], rasterSpace[k].vertices[0], p);
				float w2 = edgeFunction(rasterSpace[k].vertices[0], rasterSpace[k].vertices[1], p);
				if (w0 >= 0 && w1 >= 0 && w2 >= 0) {
					SDL_SetRenderDrawColor(gRenderer, object->tris[k].rgb.r, object->tris[k].rgb.g, object->tris[k].rgb.b, 0xff);
					SDL_RenderDrawPoint(gRenderer, p.x, p.y);
				}
			}
		}
	}
	


}

void rotateZAxis(Object_t* object, float theta) {
	for (int i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			Vec3f_t rotated;
			rotated.x = object->tris[i].vertices[j].x * cos(theta) - object->tris[i].vertices[j].y * sin(theta);
			rotated.y = object->tris[i].vertices[j].y * cos(theta) + object->tris[i].vertices[j].x * sin(theta);
			object->tris[i].vertices[j].x = rotated.x;
			object->tris[i].vertices[j].y = rotated.y;
		}
	}
}

void rotateXAxis(Object_t* object, float theta) {
	for (int i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			Vec3f_t rotated;
			rotated.y = object->tris[i].vertices[j].y * cos(theta) - object->tris[i].vertices[j].z * sin(theta);
			rotated.z = object->tris[i].vertices[j].z * cos(theta) + object->tris[i].vertices[j].y * sin(theta);
			object->tris[i].vertices[j].y = rotated.y;
			object->tris[i].vertices[j].z = rotated.z;
		}
	}
}

void rotateYAxis(Object_t* object, float theta) {
	for (int i = 0; i < object->size; i++) {
		for (int j = 0; j < 3; j++) {
			Vec3f_t rotated;
			rotated.x = object->tris[i].vertices[j].x * cos(theta) - object->tris[i].vertices[j].z * sin(theta);
			rotated.z = object->tris[i].vertices[j].z * cos(theta) + object->tris[i].vertices[j].x * sin(theta);
			object->tris[i].vertices[j].x = rotated.x;
			object->tris[i].vertices[j].z = rotated.z;
		}
	}
}



bool init() {
    bool success = true;

	if (SDL_Init(SDL_INIT_VIDEO) < 0) {
		printf("SDL could not initialize! SDL_Error: %s\n", SDL_GetError());
		success = false;
	}
	else {
		//Create window
		gWindow = SDL_CreateWindow("SDL Tutorial", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
		if (gWindow == NULL)
		{
			printf("Window could not be created! SDL_Error: %s\n", SDL_GetError());
			success = false;
		}
		else
		{
			gRenderer = SDL_CreateRenderer(gWindow, -1, SDL_RENDERER_ACCELERATED);
			if (gRenderer == NULL) {
				printf("Renderer could not be created! SDL Error: %s\n", SDL_GetError());
				success = false;
			}

		}
	}
	return success;
}

void close() {
	SDL_DestroyRenderer(gRenderer);
	SDL_DestroyWindow(gWindow);
	gWindow = NULL;
	gRenderer = NULL;

	SDL_Quit();
}
